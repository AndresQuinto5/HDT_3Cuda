#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK_RETURN(value) {           \
    hipError_t _m_cudaStat = value;         \
    if (_m_cudaStat != hipSuccess) {        \
         fprintf(stderr, "Error %s at line %d in file %s\n",              \
                 hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);    \
         exit(1);                                                         \
       } }

__global__ void vadd (int *a, int *b, int *c, int N)
{
  int myID = blockIdx.x * blockDim.x + threadIdx.x;
  if (myID < N)
    c[myID] = a[myID] + b[myID];
}

int main (void)
{
  int *ha, *hb, *hc, *da, *db, *dc;     // host (h*) and device (d*) pointers
  int i, N, BLOCK_SIZE = 256;

  printf("Enter the size of the vectors: ");
  scanf("%d", &N);

  ha = (int*)malloc(sizeof(int)*N);
  hb = (int*)malloc(sizeof(int)*N);
  hc = (int*)malloc(sizeof(int)*N);

  CUDA_CHECK_RETURN (hipMalloc ((void **) &da, sizeof (int) * N)); 
  CUDA_CHECK_RETURN (hipMalloc ((void **) &db, sizeof (int) * N));
  CUDA_CHECK_RETURN (hipMalloc ((void **) &dc, sizeof (int) * N));

  for (i = 0; i < N; i++)
  {
    ha[i] = rand () % 10000;
    hb[i] = rand () % 10000;
  }

  CUDA_CHECK_RETURN (hipMemcpy (da, ha, sizeof (int) * N, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN (hipMemcpy (db, hb, sizeof (int) * N, hipMemcpyHostToDevice));

  int grid = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  vadd <<< grid, BLOCK_SIZE >>> (da, db, dc, N);

  CUDA_CHECK_RETURN (hipDeviceSynchronize ());

  hipEventRecord(stop);

  CUDA_CHECK_RETURN (hipGetLastError ());
  CUDA_CHECK_RETURN (hipMemcpy (hc, dc, sizeof (int) * N, hipMemcpyDeviceToHost));

  for (i = 0; i < N; i++)
  {
    if (hc[i] != ha[i] + hb[i])
      printf ("Error at index %i : %i VS %i\n", i, hc[i], ha[i] + hb[i]);
  }

  CUDA_CHECK_RETURN (hipFree ((void *) da));
  CUDA_CHECK_RETURN (hipFree ((void *) db));
  CUDA_CHECK_RETURN (hipFree ((void *) dc));
  free(ha);
  free(hb);
  free(hc);

  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Time elapsed: %f ms\n", milliseconds);

  CUDA_CHECK_RETURN (hipDeviceReset ());

  return 0;
}
